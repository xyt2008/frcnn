#include "hip/hip_runtime.h"
// ------------------------------------------------------------------
// Faster R-CNN
// Copyright (c) 2015 Microsoft
// Licensed under The MIT License [see fast-rcnn/LICENSE for details]
// Written by Shaoqing Ren
// ------------------------------------------------------------------

#include "caffe/FRCNN/util/frcnn_gpu_nms.hpp"
#include <vector>
#include <iostream>

#define CUDA_CHECK(condition) \
  /* Code block avoids redefinition of hipError_t error */ \
  do { \
    hipError_t error = condition; \
    if (error != hipSuccess) { \
      std::cout << hipGetErrorString(error) << std::endl; \
    } \
  } while (0)

#define DIVUP(m,n) ((m) / (n) + ((m) % (n) > 0))
int const threadsPerBlock = sizeof(unsigned long long) * 8;

__device__ inline float devIoU(float const * const a, float const * const b) {
  float left = max(a[0], b[0]), right = min(a[2], b[2]);
  float top = max(a[1], b[1]), bottom = min(a[3], b[3]);
  float width = max(right - left + 1, 0.f), height = max(bottom - top + 1, 0.f);
  float interS = width * height;
  float Sa = (a[2] - a[0] + 1) * (a[3] - a[1] + 1);
  float Sb = (b[2] - b[0] + 1) * (b[3] - b[1] + 1);
  return interS / (Sa + Sb - interS);
}

// fyk dev_boxes dim is 4 or 5
// boxes_dim must be a const value not a var,or NVCC will not compile
#define box_dim 4

__global__ void nms_kernel(const int n_boxes, const float nms_overlap_thresh,
                           const float *dev_boxes, unsigned long long *dev_mask) {
  const int row_start = blockIdx.y;
  const int col_start = blockIdx.x;

  // if (row_start > col_start) return;

  const int row_size =
        min(n_boxes - row_start * threadsPerBlock, threadsPerBlock);
  const int col_size =
        min(n_boxes - col_start * threadsPerBlock, threadsPerBlock);

  __shared__ float block_boxes[threadsPerBlock * box_dim];
  if (threadIdx.x < col_size) {
    for (int d = 0; d < box_dim; d ++) {
      block_boxes[threadIdx.x * box_dim + d] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * box_dim + d];
    }
  }
  __syncthreads();

  if (threadIdx.x < row_size) {
    const int cur_box_idx = threadsPerBlock * row_start + threadIdx.x;
    const float *cur_box = dev_boxes + cur_box_idx * box_dim;
    int i = 0;
    unsigned long long t = 0;
    int start = 0;
    if (row_start == col_start) {
      start = threadIdx.x + 1;
    }
    for (i = start; i < col_size; i++) {
      if (devIoU(cur_box, block_boxes + i * box_dim) > nms_overlap_thresh) {
        t |= 1ULL << i;
      }
    }
    const int col_blocks = DIVUP(n_boxes, threadsPerBlock);
    dev_mask[cur_box_idx * col_blocks + col_start] = t;
  }
}

void _set_device(int device_id) {
  if (device_id<=0) return;
  int current_device;
  CUDA_CHECK(hipGetDevice(&current_device));
  if (current_device == device_id) {
    return;
  }
  // The call to hipSetDevice must come before any calls to Get, which
  // may perform initialization using the GPU.
  CUDA_CHECK(hipSetDevice(device_id));
}

namespace caffe {

namespace Frcnn {

// code from https://github.com/rbgirshick/py-faster-rcnn/blob/master/lib/nms/nms_kernel.cu
// fyk:params is all cpu memory var, boxes_dim should be 4 instead of 5(x1,y1,x2,y2,confidence),the boxes_host should have already sorted by confidence from big to little
void _nms(int* keep_out, int* num_out, const float* boxes_host, int boxes_num,
          int boxes_dim, float nms_overlap_thresh, int device_id) {

  _set_device(device_id);

  float* boxes_dev = NULL;
  unsigned long long* mask_dev = NULL;

  const int col_blocks = DIVUP(boxes_num, threadsPerBlock);

  CUDA_CHECK(hipMalloc(&boxes_dev,
                        boxes_num * boxes_dim * sizeof(float)));
  CUDA_CHECK(hipMemcpy(boxes_dev,
                        boxes_host,
                        boxes_num * boxes_dim * sizeof(float),
                        hipMemcpyHostToDevice));

  CUDA_CHECK(hipMalloc(&mask_dev,
                        boxes_num * col_blocks * sizeof(unsigned long long)));

  dim3 blocks(DIVUP(boxes_num, threadsPerBlock),
              DIVUP(boxes_num, threadsPerBlock));
  dim3 threads(threadsPerBlock);
  nms_kernel<<<blocks, threads>>>(boxes_num,
                                  nms_overlap_thresh,
                                  boxes_dev,
                                  mask_dev);

  std::vector<unsigned long long> mask_host(boxes_num * col_blocks);
  CUDA_CHECK(hipMemcpy(&mask_host[0],
                        mask_dev,
                        sizeof(unsigned long long) * boxes_num * col_blocks,
                        hipMemcpyDeviceToHost));

  std::vector<unsigned long long> remv(col_blocks);
  memset(&remv[0], 0, sizeof(unsigned long long) * col_blocks);

  int num_to_keep = 0;
  for (int i = 0; i < boxes_num; i++) {
    int nblock = i / threadsPerBlock;
    int inblock = i % threadsPerBlock;

    if (!(remv[nblock] & (1ULL << inblock))) {
      keep_out[num_to_keep++] = i;
      unsigned long long *p = &mask_host[0] + i * col_blocks;
      for (int j = nblock; j < col_blocks; j++) {
        remv[j] |= p[j];
      }
    }
  }
  *num_out = num_to_keep;

  CUDA_CHECK(hipFree(boxes_dev));
  CUDA_CHECK(hipFree(mask_dev));
}

__global__ void overlaps_kernel(const int N, const int K, const float* dev_boxes,
                           const float * dev_query_boxes, float* dev_overlaps) {
  const int row_start = blockIdx.x;
  const int col_start = blockIdx.y;
  // note that this statement is different with nms_kernel, can also asign col_start with blockIdx.x and the order doesn't matter if the following statedments is right.

  // if (row_start > col_start) return;

  const int row_size =
        min(N - row_start * threadsPerBlock, threadsPerBlock);
  const int col_size =
        min(K - col_start * threadsPerBlock, threadsPerBlock);

  __shared__ float block_boxes[threadsPerBlock * 4];
  __shared__ float block_query_boxes[threadsPerBlock * 4];
  if (threadIdx.x < col_size) {
    block_query_boxes[threadIdx.x * 4 + 0] =
        dev_query_boxes[(threadsPerBlock * col_start + threadIdx.x) * 4 + 0];
    block_query_boxes[threadIdx.x * 4 + 1] =
        dev_query_boxes[(threadsPerBlock * col_start + threadIdx.x) * 4 + 1];
    block_query_boxes[threadIdx.x * 4 + 2] =
        dev_query_boxes[(threadsPerBlock * col_start + threadIdx.x) * 4 + 2];
    block_query_boxes[threadIdx.x * 4 + 3] =
        dev_query_boxes[(threadsPerBlock * col_start + threadIdx.x) * 4 + 3];
  }
  if (threadIdx.x < row_size) {
    block_boxes[threadIdx.x * 4 + 0] =
        dev_boxes[(threadsPerBlock * row_start + threadIdx.x) * 4 + 0];
    block_boxes[threadIdx.x * 4 + 1] =
        dev_boxes[(threadsPerBlock * row_start + threadIdx.x) * 4 + 1];
    block_boxes[threadIdx.x * 4 + 2] =
        dev_boxes[(threadsPerBlock * row_start + threadIdx.x) * 4 + 2];
    block_boxes[threadIdx.x * 4 + 3] =
        dev_boxes[(threadsPerBlock * row_start + threadIdx.x) * 4 + 3];
  }
  __syncthreads();

  if (threadIdx.x < row_size) {
    for(int i = 0;i < col_size; i++) {
      int offset = row_start*threadsPerBlock * K + col_start*threadsPerBlock + threadIdx.x*K+ i ;
      dev_overlaps[offset] = devIoU(block_boxes + threadIdx.x * 4, block_query_boxes + i * 4);
    }
  }
}
// fyk:params is all cpu memory var, boxes_dim should be 4(x1,y1,x2,y2)
void _overlaps(float* overlaps,const float* boxes,const float* query_boxes, int n, int k, int device_id) {

  _set_device(device_id);

  float* overlaps_dev = NULL;
  float* boxes_dev = NULL;
  float* query_boxes_dev = NULL;

  CUDA_CHECK(hipMalloc(&boxes_dev,
                        n * 4 * sizeof(float)));

  CUDA_CHECK(hipMemcpy(boxes_dev,
                        boxes,
                        n * 4 * sizeof(float),
                        hipMemcpyHostToDevice));

  CUDA_CHECK(hipMalloc(&query_boxes_dev,
                        k * 4 * sizeof(float)));

  CUDA_CHECK(hipMemcpy(query_boxes_dev,
                        query_boxes,
                        k * 4 * sizeof(float),
                        hipMemcpyHostToDevice));

  CUDA_CHECK(hipMalloc(&overlaps_dev,
                        n * k * sizeof(float)));

  dim3 blocks(DIVUP(n, threadsPerBlock),
              DIVUP(k, threadsPerBlock));
		
  dim3 threads(threadsPerBlock);

  overlaps_kernel<<<blocks, threads>>>(n, k,
                                    boxes_dev,
                                    query_boxes_dev,
                                    overlaps_dev);  

  CUDA_CHECK(hipMemcpy(overlaps,
                        overlaps_dev,
                        n * k * sizeof(float),
                        hipMemcpyDeviceToHost));

  CUDA_CHECK(hipFree(overlaps_dev));
  CUDA_CHECK(hipFree(boxes_dev));
  CUDA_CHECK(hipFree(query_boxes_dev));

}

} // namespace frcnn

} // namespace caffe

